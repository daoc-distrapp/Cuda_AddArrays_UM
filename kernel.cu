﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__
void addArrays(int* A, int* B, int* C) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	C[i] = A[i] + B[i];
}

int main(void) {
	int N = 1024;

	int *A, *B, *C;

	// Crea los buffer, con Unified Memory, para los datos de entrada y salida
	hipMallocManaged(&A, N * sizeof(int));
	hipMallocManaged(&B, N * sizeof(int));
	hipMallocManaged(&C, N * sizeof(int));

	// Inicializa los buffer del host con los valores de entrada
	for (int i = 0; i < N; i++) {
		A[i] = i; //0,1,2,...,1023
		B[i] = N - i; //1023,1022,...,0
	}

	// Ejecuta la kernel en la GPU (4 bloques * 256 hilos = 1024 elementos calculados)
	addArrays <<<4, 256>>> (A, B, C);

	// Espera que termine la kernel
	hipDeviceSynchronize();

	// Presenta el resultado
	for (int i = 0; i < N; i++) {
		printf("Resultados %d: (%d + %d = %d)\n", i, A[i], B[i], C[i]);
	}

	// Libera los recursos
	hipFree(A);
	hipFree(B);
	hipFree(C);

}
